#include "hip/hip_runtime.h"
#include "util.hpp"
#include <random>
#include <math.h>

#include "camera.cpp"


#define DEBUGA(x) std::cout << __FILE__ << ":" << __LINE__ << " - " << #x << " = " << (x) << std::endl
#define DEBUGV(x) std::cout << (x)


#define TITLE "NBODY"
#define MAXBODYCOUNT 1024

//mandatory forward declaration
namespace wdw{
    void nbodyParam();
}
namespace gbl{
    int max_fps;
}
namespace gpu{
}

struct Body{
    float4 pos;
    float4 vel;
    float mass;
};

namespace nbd{
    Body* h_bodies1;
    Body* h_bodies2;
    Body* d_bodies1;
    Body* d_bodies2;

    float dpos = 1.0f;
    float dvel = 0.0001f;
    static float minmass = 1.0f;
    static float maxmass = 5.0f;

    void randomBodies(Body* bodies, int bodycount){
        float x, y, z, r;
        for(int i =0; i<bodycount; i++){
            Body& body = bodies[i];
            //pos
            x = (2*((rand()%1000)/1000.0f)-1);
            y = (2*((rand()%1000)/1000.0f)-1);
            z = (2*((rand()%1000)/1000.0f)-1);
            r = (rand()%1000)/1000.0f/sqrt(x*x+y*y+z*z);
            body.pos.x = r*dpos*x;
            body.pos.y = r*dpos*y;
            body.pos.z = r*dpos*z;
            body.pos.w = 1.0f;

            //vel
            x = (2*((rand()%1000)/1000.0f)-1);
            y = (2*((rand()%1000)/1000.0f)-1);
            z = (2*((rand()%1000)/1000.0f)-1);
            r = (rand()%1000)/1000.0f/sqrt(x*x+y*y+z*z);
            body.vel.x = r*dvel*x;
            body.vel.y = r*dvel*y;
            body.vel.z = r*dvel*z;
            body.vel.w = 1.0f;

            //mass
            body.mass = minmass+(maxmass-minmass)*((rand()%1000)/1000.0f);
        }
    }

    void updtBody(Body* body){

    }


}


//complexe numbe stored as a+ib
class Complex {
public:
    float a;
    float b;
    __device__ __host__ Complex(){}
    __device__ __host__ Complex(float a, float b) : a(a), b(b){}

    __device__ __host__ Complex operator+(const Complex& other) const {
        return Complex(a + other.a, b + other.b);
    }
    __device__ __host__ Complex operator*(const Complex& other) const {
        return Complex(a * other.a - b * other.b, a * other.b + b * other.a);
    }

};
struct MyCol{ //used for passing color to gpu
    float x, y, z, w;
    __device__ __host__ MyCol(){}
    __device__ __host__ MyCol(float x, float y, float z, float w) : x(x), y(y), z(z), w(w){}
    __device__ __host__ MyCol(float* c) : x(c[0]), y(c[1]), z(c[2]), w(c[3]){}
    __device__ __host__ MyCol(const MyCol& c) : x(c.x), y(c.y), z(c.z), w(c.w){}
};


struct Param {
    float scale;
    float mx, my; //mousepose
    Complex offset;

    //nbd
    int nbBodies;
};
Param h_params;
__constant__ Param d_params;



namespace cpu{
    void imp_NBody();

    void init(){
        //bugs::h_grid1 = (float4*)malloc(gbl::SCREEN_X*gbl::SCREEN_Y*sizeof(float4));
        //bugs::h_grid2 = (float4*)malloc(gbl::SCREEN_X*gbl::SCREEN_Y*sizeof(float4));
        nbd::h_bodies1 = (Body*) malloc(MAXBODYCOUNT * sizeof(Body));
        nbd::h_bodies2 = (Body*) malloc(MAXBODYCOUNT * sizeof(Body));
        gbl::display = imp_NBody;
    }
    void clean(){
        free(nbd::h_bodies1); nbd::h_bodies1 = nullptr;
        free(nbd::h_bodies2); nbd::h_bodies2 = nullptr;
    }
    void reinit(){
        //bugs::h_grid1 = (float4*)realloc(bugs::h_grid1, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4));
        //bugs::h_grid2 = (float4*)realloc(bugs::h_grid2, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4));
    }


    void imp_NBody() {
		// your N-body algorithm here!
        for (int i=0;i<h_params.nbBodies;i++)
        {
            nbd::updtBody(nbd::h_bodies1+i);
        }
	}

}//end namespace cpu

namespace gpu{
    void imp_Bugs();

    void init(){
        // checkCudaErrors( hipHostMalloc((void**) &bugs::h_grid1, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4)));
	    // checkCudaErrors( hipMalloc((void**)&bugs::d_grid1, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4)) );
	    // checkCudaErrors( hipMalloc((void**)&bugs::d_grid2, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4)) );

        //ideally transfer active grid in cpu to gpu but due to architecture not possible to fetch previous grid because it was cleaned
        //checkCudaErrors( hipMemcpy(bugs::d_grid1, bugs::h_grid, gbl::SCREEN_X*gbl::SCREEN_Y*sizeof(float4), hipMemcpyHostToDevice) );
        gbl::display = imp_Bugs;
    }
    void clean(){
        // checkCudaErrors( hipHostFree(bugs::h_grid1));
	    // checkCudaErrors( hipFree(bugs::d_grid1) );
	    // checkCudaErrors( hipFree(bugs::d_grid2) );

    }
    void reinit(){
        clean();
        init();
    }

    void setDeviceParameters(const Param& params) {
        checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(d_params), &params, sizeof(Param)) );
    }



    __global__ void kernelBugs(float4* gridOld, float4* gridNew, int SCREENX, int SCREENY) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index < SCREENX * SCREENY) {
            //access constant memory once per thread !
            Param t_params = d_params;
            //float scale = t_params.scale;
            //float sx = t_params.mx;
            //float sy = t_params.my;


            //deduce i, j (pixel coordinate) from threadIdx, blockIdx 
            int i = index / SCREENX;
		    int j = index - i * SCREENX;

			//new cell
            float4* cellOld = gridOld + (i * SCREENX + j);
            float4* cellNew = gridNew + (i * SCREENX + j);
            
		}
	}

    void imp_Bugs(){
        //initialisation
        int N = gbl::SCREEN_X * gbl::SCREEN_Y;
		int M = 256;

        //always swap from grid 1 to grid 2 and sawp pointers after

        //computation
        //kernelBugs << <(N + M - 1) / M, M >> > (bugs::d_grid1, bugs::d_grid2, gbl::SCREEN_X, gbl::SCREEN_Y);

        //fecth grid from GPU to CPU and swap grid
        //checkCudaErrors( hipMemcpy(bugs::h_grid1, bugs::d_grid2, N * sizeof(float4), hipMemcpyDeviceToHost));
        //std::swap(bugs::d_grid1, bugs::d_grid2);
    }

}//end namespace gpu




namespace wdw{
    static void HelpMarker(const char* desc){
        ImGui::TextDisabled("(?)");
        if (ImGui::BeginItemTooltip())
        {
            ImGui::PushTextWrapPos(ImGui::GetFontSize() * 35.0f);
            ImGui::TextUnformatted(desc);
            ImGui::PopTextWrapPos();
            ImGui::EndTooltip();
        }
    }

    void automataParam(){
        ImGui::Begin("Nbodies");



        ImGui::NewLine();

        ImGui::SeparatorText("Advanced parameters");
        ImGui::InputInt("max iter/frame", &gbl::max_fps);
        if(ImGui::InputInt("display count", &h_params.nbBodies)) h_params.nbBodies = h_params.nbBodies > MAXBODYCOUNT ? MAXBODYCOUNT : h_params.nbBodies;

        
       
        ImGui::SeparatorText("Options");
        if(ImGui::Button("regenerate")){
            gbl::paused = true;
            nbd::randomBodies(nbd::h_bodies1, MAXBODYCOUNT);
            if(gbl::mode == GPU_MODE) checkCudaErrors( hipMemcpy(nbd::d_bodies1, nbd::h_bodies1, MAXBODYCOUNT*sizeof(Body), hipMemcpyHostToDevice) );
            gbl::paused = false;
        }
        ImGui::DragFloat("dpos", &nbd::dpos,0.01f,0.0f,2.0f);
        ImGui::DragFloat("dvel", &nbd::dvel, 0.01f, 0.0f, 2.0f, "%.5f");
        ImGui::DragFloat("min mass", &nbd::minmass,0.5f,0.1f,10.0f);
        ImGui::DragFloat("max mass", &nbd::maxmass,0.5f,1.0f,10.0f);


        ImGui::End();
    }

    void wdw_additional(){
        ImGui::SeparatorText("GPU mode");
        static int current_gpu_mode = 0;
        const char* items[] = { "default", "shared"};

        if (ImGui::Combo("Combo", &current_gpu_mode, items, IM_ARRAYSIZE(items))) {
            switch (current_gpu_mode)
            {
            //TODO CHANGE CALLBACK
            case 0: /* gbl::display = gpu::imp_Bugs_default; */ break;
            case 1: /* gbl::display = gpu::imp_Bugs_shared; */ break;
            default: break;
            }
        }

        ImGui::SameLine(); HelpMarker(
                    "refer to readme.md for additional information\n"
                    "on how shared mode works");
    }
}//end namespace wdw




void clean(){
	switch (gbl::mode){
        case CPU_MODE: gpu::clean(); break;
        case GPU_MODE: cpu::clean(); break;
	}
}
void init(){
	switch (gbl::mode){
        case CPU_MODE: cpu::init(); break;
        case GPU_MODE: gpu::init(); break;
	}
}
void reinit(){
	switch (gbl::mode){
        case CPU_MODE: cpu::reinit(); break;
        case GPU_MODE: gpu::reinit(); break;
	}
}


namespace cbk{ 
    /*various callback
    You must ALWAYS forward the event to ImGui before processing it (except window resizing)
    You can find relevant ImGui callback in ./imgui/imgui_impl_glfw.cpp line 536 in function ImGui_ImplGlfw_InstallCallbacks
    */


    void key(GLFWwindow* window, int key, int scancode, int action, int mods){
        // Forward the event to ImGui
        ImGui_ImplGlfw_KeyCallback(window, key, scancode, action, mods);

        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureKeyboard){
            /* uses US keyboard layout ! https://www.glfw.org/docs/latest/group__keys.html
            use charCallback if you want to avoid translation qwerty->azerty*/
            if (key == GLFW_KEY_Z && action == GLFW_PRESS){ //match W in azerty
                gbl::otherWindow = !gbl::otherWindow;
            }
        }
    }

    void updt_mpos(double xpos, double ypos){
        h_params.mx = h_params.offset.a + (float)(h_params.scale*(xpos - gbl::SCREEN_X / 2));
        h_params.my = h_params.offset.b - (float)(h_params.scale*(ypos - gbl::SCREEN_Y / 2));
    }

    void mouse_button(GLFWwindow* window, int button, int action, int mods){
        // Forward the event to ImGui
        ImGui_ImplGlfw_MouseButtonCallback(window, button, action, mods);
        
        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            double xpos, ypos;
            glfwGetCursorPos(window, &xpos, &ypos);
            if(button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS){
                updt_mpos(xpos, ypos);
            }
            if(button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_PRESS){
                h_params.offset.a += (float)(h_params.scale * (xpos - gbl::SCREEN_X / 2));
		        h_params.offset.b += -(float)(h_params.scale * (ypos - gbl::SCREEN_Y / 2));
            }
        }
    }

    void cursor_position(GLFWwindow* window, double xpos, double ypos){
        //forward the event to ImGui
        ImGui_ImplGlfw_CursorPosCallback(window, xpos, ypos);

        //if ImGui doesn't want the event, process i
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            int leftState = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT);
            if(leftState == GLFW_PRESS){
                updt_mpos(xpos, ypos);
            }
        }
    }

    void scroll(GLFWwindow* window, double xoffset, double yoffset){
        // Forward the event to ImGui
        ImGui_ImplGlfw_ScrollCallback(window, xoffset, yoffset);
        
        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            float fac = (GLFW_PRESS == glfwGetKey(window, GLFW_KEY_LEFT_CONTROL)) ?  1.16f : 1.05f;
            if (yoffset >0) h_params.scale /= fac;
	        else h_params.scale *= fac;
        }
    }

    void window_size(GLFWwindow* window, int width, int height){
        //reszing logic handled in gbl::resizePixelsBuffer() called from gbl::calculate
        gbl::paused = true;
        gbl::needResize = true;
    }

}//end namespace cbk

int main(void){
    

    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(gbl::SCREEN_X, gbl::SCREEN_Y, TITLE, NULL, NULL);
    if (!window){
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    utl::initImGui(window);

    /* malloc values ...*/
    init();
    {/* set up generic parameters*/
        h_params.scale = 0.003f;
        h_params.mx = 0.0f;
        h_params.my = 0.0f;
        h_params.offset = Complex(0.0f, 0.0f);
        


        //framerate
        gbl::max_fps = 20;
        

        //nbd
        h_params.nbBodies = 32;

        glClearColor(0.0,0.0,0.0,0.0);
        glColor4f(1.0,1.0,1.0,1.0);
        glDisable(GL_DEPTH_TEST);
        glPointSize(2.0f);

        nbd::randomBodies(nbd::h_bodies1, MAXBODYCOUNT);
    }

    /* Initialize callback*/
    glfwSetKeyCallback(window, cbk::key);
    glfwSetMouseButtonCallback(window, cbk::mouse_button);
    glfwSetCursorPosCallback(window, cbk::cursor_position);
    glfwSetScrollCallback(window, cbk::scroll);
    glfwSetWindowSizeCallback(window, cbk::window_size);


    /*start timer*/
    double last_frame_time = glfwGetTime();

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Poll for and process events */
        glfwPollEvents();

        /* Interface*/
        utl::newframeImGui();
        if(gbl::otherWindow) {
            utl::wdw_info(gbl::mode, gbl::SCREEN_X,gbl::SCREEN_Y,gbl::currentFPS);
            wdw::automataParam();
        }
        
        //timer management
        double curr_time = glfwGetTime();
        double framerate = (double)1 / (double)gbl::max_fps;


        /* Render */
        gbl::calculate(window);
        gpu::setDeviceParameters(h_params);
        if(curr_time -  last_frame_time > framerate){
            gbl::display();
            last_frame_time = curr_time;
        }        
        if(!gbl::paused){
            cameraApply();
            glClear(GL_COLOR_BUFFER_BIT);
            glEnableClientState(GL_VERTEX_ARRAY);
            glVertexPointer(4, GL_FLOAT, sizeof(Body), &(nbd::h_bodies1->pos));
	        glDrawArrays(GL_POINTS, 0, h_params.nbBodies);
	        glDisableClientState(GL_VERTEX_ARRAY);
        }
  

        /* end frame for imgui*/
        utl::endframeImGui();
        utl::multiViewportImGui(window);
        

        /* Swap front and back buffers */
        glfwSwapBuffers(window);
    }

    utl::shutdownImGui();
    glfwTerminate();
    return 0;
}