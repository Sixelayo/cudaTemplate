#include "hip/hip_runtime.h"
#include "util.hpp"
#include <random>
#include <math.h>

#include "camera.cpp"


#define DEBUGA(x) std::cout << __FILE__ << ":" << __LINE__ << " - " << #x << " = " << (x) << std::endl
#define DEBUGV(x) std::cout << (x)


#define TITLE "NBODY"


//mandatory forward declaration
namespace wdw{
    void nbodyParam();
}
namespace gbl{
    int max_fps;
}
namespace gpu{
}

struct Body{
    float3 pos; 
    float3 vel;
    float mass;
};

namespace nbd{
    Body* h_bodies1;
    Body* h_bodies2;
    Body* d_bodies1;
    Body* d_bodies2;

    static int MAXBODYCOUNT = 1024;
    float dpos = 2.5f;
    float dvel = 0.0001f;
    static float minmass = 1.0f;
    static float maxmass = 5.0f;


    void randomBodies(Body* bodies, int bodycount){
        float x, y, z, r;
        for(int i =0; i<bodycount; i++){
            Body& body = bodies[i];
            //pos
            x = (2*((rand()%1000)/1000.0f)-1);
            y = (2*((rand()%1000)/1000.0f)-1);
            z = (2*((rand()%1000)/1000.0f)-1);
            r = (rand()%1000)/1000.0f/sqrt(x*x+y*y+z*z);
            body.pos.x = r*dpos*x;
            body.pos.y = r*dpos*y;
            body.pos.z = r*dpos*z;
            //body.pos.w = 1.0f;

            //vel
            x = (2*((rand()%1000)/1000.0f)-1);
            y = (2*((rand()%1000)/1000.0f)-1);
            z = (2*((rand()%1000)/1000.0f)-1);
            r = (rand()%1000)/1000.0f/sqrt(x*x+y*y+z*z);
            body.vel.x = r*dvel*x;
            body.vel.y = r*dvel*y;
            body.vel.z = r*dvel*z;
            //body.vel.w = 1.0f;

            //mass
            body.mass = minmass+(maxmass-minmass)*((rand()%1000)/1000.0f);
        }
    }

    __host__ __device__ float3 add(const float3& a, const float3& b) {
        return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
    }
    __host__ __device__ float3 minus(const float3& a, const float3& b) {
        return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
    }
    __host__ __device__ float3 multiply(float s, const float3& v) {
        return make_float3(s * v.x, s * v.y, s * v.z);
    }

    __host__ __device__ inline float dist(const Body& b1, const Body& b2){
        return sqrtf(   (b1.pos.x - b2.pos.x) * (b1.pos.x - b2.pos.x) +
                        (b1.pos.y - b2.pos.y) * (b1.pos.y - b2.pos.y) +
                        (b1.pos.z - b2.pos.z) * (b1.pos.z - b2.pos.z));
    }
    __host__ __device__ inline float length2(float3 v) {
        return v.x * v.x + v.y * v.y + v.z * v.z;
    }



}


//complexe numbe stored as a+ib
class Complex {
public:
    float a;
    float b;
    __device__ __host__ Complex(){}
    __device__ __host__ Complex(float a, float b) : a(a), b(b){}

    __device__ __host__ Complex operator+(const Complex& other) const {
        return Complex(a + other.a, b + other.b);
    }
    __device__ __host__ Complex operator*(const Complex& other) const {
        return Complex(a * other.a - b * other.b, a * other.b + b * other.a);
    }

};
struct MyCol{ //used for passing color to gpu
    float x, y, z, w;
    __device__ __host__ MyCol(){}
    __device__ __host__ MyCol(float x, float y, float z, float w) : x(x), y(y), z(z), w(w){}
    __device__ __host__ MyCol(float* c) : x(c[0]), y(c[1]), z(c[2]), w(c[3]){}
    __device__ __host__ MyCol(const MyCol& c) : x(c.x), y(c.y), z(c.z), w(c.w){}
};


struct Param {
    float scale;
    float mx, my; //mousepose
    Complex offset;

    //nbd
    int nbBodies; //nb à afficher
    float G;
    float EPS2;
};
Param h_params;
__constant__ Param d_params;



namespace cpu{
    void imp_NBody();

    void init(){
        nbd::h_bodies1 = (Body*) malloc(nbd::MAXBODYCOUNT * sizeof(Body));
        nbd::h_bodies2 = (Body*) malloc(nbd::MAXBODYCOUNT * sizeof(Body));
        gbl::display = imp_NBody;
    }
    void clean(){
        free(nbd::h_bodies1); nbd::h_bodies1 = nullptr;
        free(nbd::h_bodies2); nbd::h_bodies2 = nullptr;
    }
    void reinit(){
        nbd::h_bodies1 = (Body*)realloc(nbd::h_bodies1, nbd::MAXBODYCOUNT * sizeof(Body));
        nbd::h_bodies2 = (Body*)realloc(nbd::h_bodies2, nbd::MAXBODYCOUNT * sizeof(Body));
    }


    void imp_NBody() {
		// your N-body algorithm here!
        for (int i=0;i<h_params.nbBodies;i++){
            float3 acc = {0.0f, 0.0f, 0.0f};

            for (int j = 0; j < h_params.nbBodies; j++) {
                if (i != j) { 
                    float3 r = nbd::minus(nbd::h_bodies1[j].pos, nbd::h_bodies1[i].pos);
                    float d = nbd::length2(r) + h_params.EPS2;  
                    float factor = h_params.G * nbd::h_bodies1[j].mass / sqrtf(d * d * d);
                    acc = nbd::add(acc, nbd::multiply(factor, r));
                }
            }
            nbd::h_bodies2[i].pos = nbd::add(nbd::h_bodies1[i].pos, nbd::h_bodies1[i].vel);
            nbd::h_bodies2[i].vel = nbd::add(nbd::h_bodies1[i].vel, acc);
        }

        std::swap(nbd::h_bodies1, nbd::h_bodies2);
	}

}//end namespace cpu

namespace gpu{
    void imp_NBody();

    void init(){
        checkCudaErrors( hipHostMalloc((void**) &nbd::h_bodies1, nbd::MAXBODYCOUNT * sizeof(Body)) );
	    checkCudaErrors( hipMalloc((void**)&nbd::d_bodies1, nbd::MAXBODYCOUNT * sizeof(Body)) );
	    checkCudaErrors( hipMalloc((void**)&nbd::d_bodies2, nbd::MAXBODYCOUNT * sizeof(Body)) );

        gbl::display = imp_NBody;
    }
    void clean(){
        checkCudaErrors( hipHostFree(nbd::h_bodies1));
	    checkCudaErrors( hipFree(nbd::d_bodies1) );
	    checkCudaErrors( hipFree(nbd::d_bodies2) );

    }
    void reinit(){
        clean();
        init();
    }

    void setDeviceParameters(const Param& params) {
        checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(d_params), &params, sizeof(Param)) );
    }



    __global__ void kernelNbody(Body* oldBodies, Body* newBodies) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index < d_params.nbBodies) {
            //access constant memory once per thread !
            //Param t_params = d_params;
            //float scale = t_params.scale;
            //float sx = t_params.mx;
            //float sy = t_params.my;

            int i = index;
            float3 acc = {0.0f, 0.0f, 0.0f};

            for (int j = 0; j < d_params.nbBodies; j++) {
                if (i != j) { 
                    float3 r = nbd::minus(oldBodies[j].pos, oldBodies[i].pos);
                    float d = nbd::length2(r) + d_params.EPS2;  
                    float factor = d_params.G * oldBodies[j].mass / sqrtf(d * d * d);
                    acc = nbd::add(acc, nbd::multiply(factor, r));
                }
            }
            newBodies[i].pos = nbd::add(oldBodies[i].pos, oldBodies[i].vel);
            newBodies[i].vel = nbd::add(oldBodies[i].vel, acc);            
		}
	}

    void imp_NBody(){
        //initialisation
        int N = h_params.nbBodies;
		int M = 256;

        //computation
        kernelNbody << <(N + M - 1) / M, M >> > (nbd::d_bodies1, nbd::d_bodies2);
        checkKernelErrors();

        //fecth newly computed bodies from GPU to CPU and swap grid
        checkCudaErrors( hipMemcpy(nbd::h_bodies1, nbd::d_bodies2, N * sizeof(Body), hipMemcpyDeviceToHost));
        std::swap(nbd::d_bodies1, nbd::d_bodies2);
    }

}//end namespace gpu




namespace wdw{
    static void HelpMarker(const char* desc){
        ImGui::TextDisabled("(?)");
        if (ImGui::BeginItemTooltip())
        {
            ImGui::PushTextWrapPos(ImGui::GetFontSize() * 35.0f);
            ImGui::TextUnformatted(desc);
            ImGui::PopTextWrapPos();
            ImGui::EndTooltip();
        }
    }

    void automataParam(){
        ImGui::Begin("Nbodies");

        ImGui::SeparatorText("Advanced parameters");
        ImGui::InputFloat("gravitaional const", &h_params.G,0,0,"%.7f");
        ImGui::InputFloat("EPSILON²", &h_params.EPS2);
        ImGui::DragFloat("min mass", &nbd::minmass,0.5f,0.1f,10.0f);
        ImGui::DragFloat("max mass", &nbd::maxmass,0.5f,1.0f,10.0f);

 

        ImGui::SeparatorText("Advanced parameters");
        ImGui::InputInt("max iter/frame", &gbl::max_fps);
        static int buffermaxcount = 1024;
        ImGui::InputInt("loaded", &buffermaxcount);
        ImGui::SameLine(); HelpMarker(
                "The number of Bodies loaded in memory\n"
                "press apply to reload");
        ImGui::SameLine();
        if(ImGui::Button("apply")){
            gbl::paused = true;
            nbd::MAXBODYCOUNT = buffermaxcount;
            h_params.nbBodies = h_params.nbBodies > nbd::MAXBODYCOUNT ? nbd::MAXBODYCOUNT : h_params.nbBodies;
            reinit();
            nbd::randomBodies(nbd::h_bodies1, nbd::MAXBODYCOUNT);
            if(gbl::mode == GPU_MODE) checkCudaErrors( hipMemcpy(nbd::d_bodies1, nbd::h_bodies1, nbd::MAXBODYCOUNT*sizeof(Body), hipMemcpyHostToDevice) );
            gbl::paused = false;
        }
        //if(ImGui::InputInt("display count", &h_params.nbBodies)) h_params.nbBodies = h_params.nbBodies > nbd::MAXBODYCOUNT ? nbd::MAXBODYCOUNT : h_params.nbBodies;
        ImGui::SliderInt("displayed", &h_params.nbBodies, 1, nbd::MAXBODYCOUNT);
        ImGui::SameLine(); HelpMarker(
                "The number of Bodies processed\n"
                "(bodies hidden aren't evalueated when\n"
                "updating positions)");

        ImGui::SeparatorText("Options");
        if(ImGui::Button("regenerate")){
            gbl::paused = true;
            nbd::randomBodies(nbd::h_bodies1, nbd::MAXBODYCOUNT);
            if(gbl::mode == GPU_MODE) checkCudaErrors( hipMemcpy(nbd::d_bodies1, nbd::h_bodies1, nbd::MAXBODYCOUNT*sizeof(Body), hipMemcpyHostToDevice) );
            gbl::paused = false;
        }
        ImGui::DragFloat("dpos", &nbd::dpos, 0.01f, 1.0f, 5.0f, "%.5f");
        ImGui::DragFloat("dvel", &nbd::dvel, 0.01f, 0.0f, 1.0f, "%.5f");



        ImGui::End();
    }

    void wdw_additional(){
        ImGui::SeparatorText("GPU mode");
        static int current_gpu_mode = 0;
        const char* items[] = { "default", "shared"};

        if (ImGui::Combo("Combo", &current_gpu_mode, items, IM_ARRAYSIZE(items))) {
            switch (current_gpu_mode)
            {
            //TODO CHANGE CALLBACK
            case 0: /* gbl::display = gpu::imp_Bugs_default; */ break;
            case 1: /* gbl::display = gpu::imp_Bugs_shared; */ break;
            default: break;
            }
        }

        ImGui::SameLine(); HelpMarker(
                    "refer to readme.md for additional information\n"
                    "on how shared mode works");
    }
}//end namespace wdw




void clean(){
	switch (gbl::mode){
        case CPU_MODE: gpu::clean(); break;
        case GPU_MODE: cpu::clean(); break;
	}
}
void init(){
	switch (gbl::mode){
        case CPU_MODE: cpu::init(); break;
        case GPU_MODE: gpu::init(); break;
	}
    nbd::randomBodies(nbd::h_bodies1, nbd::MAXBODYCOUNT);
    if(gbl::mode == GPU_MODE) checkCudaErrors( hipMemcpy(nbd::d_bodies1, nbd::h_bodies1, nbd::MAXBODYCOUNT*sizeof(Body), hipMemcpyHostToDevice) );
}
void reinit(){
    hipDeviceSynchronize();
	switch (gbl::mode){
        case CPU_MODE: cpu::reinit(); break;
        case GPU_MODE: gpu::reinit(); break;
	}
    hipDeviceSynchronize();
}


namespace cbk{ 
    /*various callback
    You must ALWAYS forward the event to ImGui before processing it (except window resizing)
    You can find relevant ImGui callback in ./imgui/imgui_impl_glfw.cpp line 536 in function ImGui_ImplGlfw_InstallCallbacks
    */


    void key(GLFWwindow* window, int key, int scancode, int action, int mods){
        // Forward the event to ImGui
        ImGui_ImplGlfw_KeyCallback(window, key, scancode, action, mods);

        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureKeyboard){
            /* uses US keyboard layout ! https://www.glfw.org/docs/latest/group__keys.html
            use charCallback if you want to avoid translation qwerty->azerty*/
            if (key == GLFW_KEY_Z && action == GLFW_PRESS){ //match W in azerty
                gbl::otherWindow = !gbl::otherWindow;
            }
        }
    }

    void updt_mpos(double xpos, double ypos){
        h_params.mx = h_params.offset.a + (float)(h_params.scale*(xpos - gbl::SCREEN_X / 2));
        h_params.my = h_params.offset.b - (float)(h_params.scale*(ypos - gbl::SCREEN_Y / 2));
    }

    void mouse_button(GLFWwindow* window, int button, int action, int mods){
        // Forward the event to ImGui
        ImGui_ImplGlfw_MouseButtonCallback(window, button, action, mods);
        
        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            double xpos, ypos;
            glfwGetCursorPos(window, &xpos, &ypos);
            if(button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS){
                updt_mpos(xpos, ypos);
            }
            if(button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_PRESS){
                h_params.offset.a += (float)(h_params.scale * (xpos - gbl::SCREEN_X / 2));
		        h_params.offset.b += -(float)(h_params.scale * (ypos - gbl::SCREEN_Y / 2));
            }
        }
    }

    void cursor_position(GLFWwindow* window, double xpos, double ypos){
        //forward the event to ImGui
        ImGui_ImplGlfw_CursorPosCallback(window, xpos, ypos);

        //if ImGui doesn't want the event, process i
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            int leftState = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT);
            if(leftState == GLFW_PRESS){
                updt_mpos(xpos, ypos);
            }
        }
    }

    void scroll(GLFWwindow* window, double xoffset, double yoffset){
        // Forward the event to ImGui
        ImGui_ImplGlfw_ScrollCallback(window, xoffset, yoffset);
        
        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            float fac = (GLFW_PRESS == glfwGetKey(window, GLFW_KEY_LEFT_CONTROL)) ?  1.16f : 1.05f;
            if (yoffset >0) h_params.scale /= fac;
	        else h_params.scale *= fac;
        }
    }

    void window_size(GLFWwindow* window, int width, int height){
        //reszing logic handled in gbl::resizePixelsBuffer() called from gbl::calculate
        gbl::paused = true;
        gbl::needResize = true;
    }

}//end namespace cbk

int main(void){
    

    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(gbl::SCREEN_X, gbl::SCREEN_Y, TITLE, NULL, NULL);
    if (!window){
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    utl::initImGui(window);

    /* malloc values ...*/
    init();
    {/* set up generic parameters*/
        h_params.scale = 0.003f;
        h_params.mx = 0.0f;
        h_params.my = 0.0f;
        h_params.offset = Complex(0.0f, 0.0f);
        


        //framerate
        gbl::max_fps = 60;
        

        //nbd
        h_params.nbBodies = 32;
        h_params.G = 0.0000001f;
        h_params.EPS2 = 0.1f;

        glClearColor(0.3,0.3,0.3,1.0);
        glColor4f(1.0,1.0,1.0,1.0);
        glDisable(GL_DEPTH_TEST);
        glPointSize(2.0f);

        nbd::randomBodies(nbd::h_bodies1, nbd::MAXBODYCOUNT);
    }

    /* Initialize callback*/
    glfwSetKeyCallback(window, cbk::key);
    glfwSetMouseButtonCallback(window, cbk::mouse_button);
    glfwSetCursorPosCallback(window, cbk::cursor_position);
    glfwSetScrollCallback(window, cbk::scroll);
    glfwSetWindowSizeCallback(window, cbk::window_size);


    /*start timer*/
    double last_frame_time = glfwGetTime();

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Poll for and process events */
        glfwPollEvents();

        /* Interface*/
        utl::newframeImGui();
        if(gbl::otherWindow) {
            utl::wdw_info(gbl::mode, gbl::SCREEN_X,gbl::SCREEN_Y,gbl::currentFPS);
            wdw::automataParam();
        }
        
        //timer management
        double curr_time = glfwGetTime();
        double framerate = (double)1 / (double)gbl::max_fps;


        /* Render */
        gbl::calculate(window);
        gpu::setDeviceParameters(h_params);
        if(curr_time -  last_frame_time > framerate){
            gbl::display();
            last_frame_time = curr_time;
        }        
        if(!gbl::paused){
            cameraApply();
            glClear(GL_COLOR_BUFFER_BIT);
            glEnableClientState(GL_VERTEX_ARRAY);
            glEnableClientState(GL_COLOR_ARRAY);
            glVertexPointer(3, GL_FLOAT, sizeof(Body), &(nbd::h_bodies1->pos));
            glColorPointer(3, GL_FLOAT, sizeof(Body), &(nbd::h_bodies1->pos)); //todo replace with color
	        glDrawArrays(GL_POINTS, 0, h_params.nbBodies);
            glDisableClientState(GL_COLOR_ARRAY);
	        glDisableClientState(GL_VERTEX_ARRAY);
        }
  

        /* end frame for imgui*/
        utl::endframeImGui();
        utl::multiViewportImGui(window);
        

        /* Swap front and back buffers */
        glfwSwapBuffers(window);
    }

    utl::shutdownImGui();
    glfwTerminate();
    return 0;
}