#include "hip/hip_runtime.h"
#include "util.hpp"
#include <random>
#include <math.h>

#include "camera.cpp"

#define INF 2e10f
#define DEBUGA(x) std::cout << __FILE__ << ":" << __LINE__ << " - " << #x << " = " << (x) << std::endl
#define DEBUGV(x) std::cout << (x)


#define TITLE "KMEANS"


//mandatory forward declaration
namespace wdw{
    void kmeansParam();
}
namespace gbl{
    int max_fps;
}
namespace gpu{
}

//complexe numbe stored as a+ib
class Complex {
public:
    float a;
    float b;
    __device__ __host__ Complex(){}
    __device__ __host__ Complex(float a, float b) : a(a), b(b){}

    __device__ __host__ Complex operator+(const Complex& other) const {
        return Complex(a + other.a, b + other.b);
    }
    __device__ __host__ Complex operator*(const Complex& other) const {
        return Complex(a * other.a - b * other.b, a * other.b + b * other.a);
    }

};
struct MyCol{ //used for passing color to gpu for type compatibility reasons
    float x, y, z, w;
    __device__ __host__ MyCol(){}
    __device__ __host__ MyCol(float x, float y, float z, float w) : x(x), y(y), z(z), w(w){}
    __device__ __host__ MyCol(float* c) : x(c[0]), y(c[1]), z(c[2]), w(c[3]){}
    __device__ __host__ MyCol(const MyCol& c) : x(c.x), y(c.y), z(c.z), w(c.w){}
};

struct Point{
    float3 pos;  //point position
    float3 col; //the point color (same of the cluster)
    int label; //which clust the point belongs to
};

namespace kmn{
    Point* h_points;
    Point* h_centroids1;
    Point* h_centroids2;
    Point* d_points;
    Point* d_centroids1;
    Point* d_centroids2;

    //value linked with buffer in wdw with buffer for chagnes
    static int NBPOINTS = 16*1024;
    static int NBCENTROIDS = 128;

    float dpoints = 1.0f;

    float3 randomColor()    {
        float3 color;
        color.x = (rand() % 1000) / 1000.0f;
        color.y = (rand() % 1000) / 1000.0f;
        color.z = (rand() % 1000) / 1000.0f;
        return color;
    }

    void randomPoints(){
        // create random points. Default colors white belonging to cluster -1 (none)
        // create artificial clusters

        //plug parm to function tempalte
        int n = NBPOINTS;
        int nbClusters = NBCENTROIDS;
        float d = dpoints;

        int i = 0;
        float x, y, z, r;
        float4* c = (float4*)malloc(nbClusters*sizeof(float4));
        float* s = (float*)malloc(nbClusters*sizeof(float));
        for (i = 0; i<nbClusters; i++)
        {
            x = (2 * ((rand() % 1000) / 1000.0f) - 1);
            y = (2 * ((rand() % 1000) / 1000.0f) - 1);
            z = (2 * ((rand() % 1000) / 1000.0f) - 1);
            r = powf(3 * (rand() % 1000) / 1000.0f, 4);

            c[i].x = r*d*x;
            c[i].y = r*d*y;
            c[i].z = r*d*z;
            c[i].w = 1.0f; // must be 1.0

            s[i] = (rand() % 1000) / 1000.0f + 0.5f;
        }

        //float4* a = (float4*)malloc(n*sizeof(float4));
        for (i = 0; i<n; i++)
        {
            int cl = rand() % NBCENTROIDS;
            x = (2 * ((rand() % 1000) / 1000.0f) - 1);
            y = (2 * ((rand() % 1000) / 1000.0f) - 1);
            z = (2 * ((rand() % 1000) / 1000.0f) - 1);
            r = powf(2 * (rand() % 1000) / 1000.0f / sqrt(x*x + y*y + z*z), 2.5);

            h_points[i].pos.x = c[cl].x + s[cl] * s[cl] * r*d*x;
            h_points[i].pos.y = c[cl].y + s[cl] * s[cl] * r*d*y;
            h_points[i].pos.z = c[cl].z + s[cl] * s[cl] * r*d*z;

            h_points[i].col = {1.0f,1.0f,1.0f}; //color set to white at first
            h_points[i].label = -1; //label set to -1 at first doesn't belong to any cluster

        }
        free(c);
        free(s);
    }
    void randomClusters(){
        for(int i=0; i < kmn::NBCENTROIDS; i++){
            kmn::h_centroids1[i].pos = kmn::h_points[i].pos;
            
            
            //reset here before swapping
            kmn::h_centroids2[i].pos = {0.0f,0.0f,0.0f}; 
            kmn::h_centroids2[i].label = 0; //used a size

            float3 c= randomColor();

            //both centroid share the same color
            kmn::h_centroids1[i].col = c;
            kmn::h_centroids2[i].col = c; //both centroid share the same colors
            
        }
    }

    __host__ __device__ float3 add(const float3& a, const float3& b) {
        return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
    }
    __host__ __device__ float3 minus(const float3& a, const float3& b) {
        return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
    }
    __host__ __device__ float3 multiply(float s, const float3& v) {
        return make_float3(s * v.x, s * v.y, s * v.z);
    }

    __host__ __device__ inline float length2(float3 v) {
        return v.x * v.x + v.y * v.y + v.z * v.z;
    }



}



struct Param {
    float scale;
    float mx, my; //mousepose
    Complex offset;

    //kmn
};
Param h_params;
__constant__ Param d_params;


inline void sendPointsToGPU(){
    checkCudaErrors( hipMemcpy(kmn::d_points, kmn::h_points, kmn::NBPOINTS*sizeof(Point), hipMemcpyHostToDevice) );
}
inline void sendCentroToGpu(){
    checkCudaErrors( hipMemcpy(kmn::d_centroids1, kmn::h_centroids1, kmn::NBCENTROIDS*sizeof(Point), hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(kmn::d_centroids2, kmn::h_centroids1, kmn::NBCENTROIDS*sizeof(Point), hipMemcpyHostToDevice) ); //(because they need color)
}
inline void getPointsFromGPU(){
    checkCudaErrors( hipMemcpy(kmn::h_points, kmn::d_points, kmn::NBPOINTS * sizeof(Point), hipMemcpyDeviceToHost));
}
inline void getCentroFromGPU(){
    checkCudaErrors( hipMemcpy(kmn::h_centroids1, kmn::d_centroids2, kmn::NBCENTROIDS * sizeof(Point), hipMemcpyDeviceToHost));
}

namespace cpu{
    void imp_KMeans();

    void init(){
        kmn::h_points = (Point*) malloc(kmn::NBPOINTS * sizeof(Point));
        kmn::h_centroids1 = (Point*) malloc(kmn::NBCENTROIDS * sizeof(Point));
        kmn::h_centroids2 = (Point*) malloc(kmn::NBCENTROIDS * sizeof(Point));
        gbl::display = imp_KMeans;
    }
    void clean(){
        free(kmn::h_points); kmn::h_points = nullptr;
        free(kmn::h_centroids1); kmn::h_centroids1 = nullptr;
        free(kmn::h_centroids2); kmn::h_centroids2 = nullptr;
    }
    void reinit(){
        kmn::h_points = (Point*)realloc(kmn::h_points, kmn::NBPOINTS * sizeof(Point));
        kmn::h_centroids1 = (Point*)realloc(kmn::h_centroids1, kmn::NBCENTROIDS * sizeof(Point));
        kmn::h_centroids2 = (Point*)realloc(kmn::h_centroids2, kmn::NBCENTROIDS * sizeof(Point));
    }

    void phase1(){
        //phase 1 assignment (assign each point to closest centroid)
        for (int i = 0; i<kmn::NBPOINTS; i++){
            float dmin = INF;
            int n = 0;
            for(int j=0; j < kmn::NBCENTROIDS; j++){
                float distance = sqrtf(kmn::length2(kmn::minus(kmn::h_centroids1[j].pos, kmn::h_points[i].pos)));
                if(distance<dmin) {
                    dmin = distance;
                    n=j;
                }
            }
            //point i assigned to closest cluster n (label and color)
            kmn::h_points[i].label = n;
            kmn::h_points[i].col = kmn::h_centroids1[n].col;
        }
    }

    void phase2(){
        //reduction, recompute centroids

        //reset new centroids
        //this operation is done once at initialisation and at at the end of phase2 right before swapping, reset centroids1 (so no need to reset here)
        // for(int j=0; j < kmn::NBCENTROIDS; j++){
        //     kmn::h_centroids2[j].pos = {0.0f,0.0f,0.0f};
        //     kmn::h_centroids2[j].label = 0; //used a size
        // }

        //for each point, add its coordinate and one to the centroids (we use label as size for centroid)
        for (int i = 0; i<kmn::NBPOINTS; i++){
            int index = kmn::h_points[i].label;
            kmn::h_centroids2[index].pos =  kmn::add(kmn::h_centroids2[index].pos, kmn::h_points[i].pos);
            kmn::h_centroids2[index].label += 1;
        }

        //divided each centroid pos by its count AND reset for next step
        for(int j=0; j < kmn::NBCENTROIDS; j++){
            kmn::h_centroids2[j].pos = kmn::multiply((float)1/kmn::h_centroids2[j].label,kmn::h_centroids2[j].pos);

            //optimization : reset here at the same time before swapping
            kmn::h_centroids1[j].pos = {0.0f,0.0f,0.0f};
            kmn::h_centroids1[j].label = 0; //used a size
        }


    }

    void imp_KMeans() {
        phase1();
        phase2();
        std::swap(kmn::h_centroids1, kmn::h_centroids2);
	}

}//end namespace cpu

namespace gpu{
    void (*gpu_cbk)();
    void imp_KmeansV1();
    void imp_KmeansV2();

    void init(){
        checkCudaErrors( hipHostMalloc((void**) &kmn::h_points, kmn::NBPOINTS * sizeof(Point)) );
        checkCudaErrors( hipMalloc((void**) &kmn::d_points, kmn::NBPOINTS * sizeof(Point)) );
        checkCudaErrors( hipHostMalloc((void**) &kmn::h_centroids1, kmn::NBCENTROIDS * sizeof(Point)) );
        checkCudaErrors( hipHostMalloc((void**) &kmn::h_centroids2, kmn::NBCENTROIDS * sizeof(Point)) ); //nescessary for phase 2 when done on cpu
        checkCudaErrors( hipMalloc((void**) &kmn::d_centroids1, kmn::NBCENTROIDS * sizeof(Point)) );
        checkCudaErrors( hipMalloc((void**) &kmn::d_centroids2, kmn::NBCENTROIDS * sizeof(Point)) ); //nescerray for phase 2 when done on gpu
        gbl::display = gpu_cbk; //uses intermediate gpu cbk for saving mode when switching back to cpu
    }
    void clean(){
        checkCudaErrors( hipHostFree(kmn::h_points));
        checkCudaErrors( hipHostFree(kmn::h_centroids1));
	    checkCudaErrors( hipFree(kmn::d_centroids1) );
	    checkCudaErrors( hipFree(kmn::d_centroids2) );
	    checkCudaErrors( hipFree(kmn::d_points) );
    }
    void reinit(){
        clean();
        init();
    }

    void setDeviceParameters(const Param& params) {
        checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(d_params), &params, sizeof(Param)) );
    }



    __global__ void kernelAssign(Point* pts, Point* oldCentro, int nbpts, int nbcentro) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index < nbpts) {
            //access constant memory once per thread !
            //Param t_params = d_params;
            //float scale = t_params.scale;
            //float sx = t_params.mx;
            //float sy = t_params.my;

            float dmin = INF;
            int n = 0;
            for(int j=0; j < nbcentro; j++){
                float distance = sqrtf(kmn::length2(kmn::minus(oldCentro[j].pos, pts[index].pos)));
                if(distance<dmin) {
                    dmin = distance;
                    n=j;
                }
            }
            //point i assigned to closest cluster n (label and color)
            pts[index].label = n;
            pts[index].col = oldCentro[n].col;
		}
	}
    __global__ void kernelReduce(Point* pts, Point* newCentro, int nbpts, int nbcentro) {
        //warning : harcodec values for block of size 256
        const int BDIM = 256;
        __shared__ Point shared_pts[BDIM]; //the 256 points preload in shared memory
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if (index < nbcentro) {
            //reset
            newCentro[index].pos = {0.0f,0.0f,0.0f};
            newCentro[index].label = 0; //used a size

            //used shared memory batch preload
            int loadIndex =0;
            unsigned int tid = threadIdx.x;
            while(loadIndex < nbpts){
                if(tid+loadIndex < nbpts ) shared_pts[tid] = pts[loadIndex+tid];
                else pts[tid] = {0.0f,0.0f,0.0f};
                __syncthreads();

                //sum partial closest points
                for (int j = 0; j < BDIM; j++) {
                    int p_index =  shared_pts[tid].label;
                    if(p_index == index){//very suboptimal
                        newCentro[index].pos =  kmn::add(newCentro[index].pos, shared_pts[tid].pos);
                        newCentro[index].label += 1;
                    }
                }
                loadIndex += BDIM;
            }

            //divided each centroid pos by its count
            newCentro[index].pos = kmn::multiply((float)1/newCentro[index].label,newCentro[index].pos);

            //optimization for later: reset here at the same time before swapping
            //oldCentro[index].pos = {0.0f,0.0f,0.0f};
            //oldCentro[index].label = 0; //used a size
        }
    }


    void imp_KmeansV1(){
        //initialisation
        int N = kmn::NBPOINTS;
		int M = 256;

        // phase1 assignment
        kernelAssign << <(N + M - 1) / M, M >> > (kmn::d_points, kmn::d_centroids1, kmn::NBPOINTS, kmn::NBCENTROIDS);
        checkKernelErrors();

        getPointsFromGPU(); //fetch label

        //phase 2 reduction
        cpu::phase2();

        std::swap(kmn::h_centroids1, kmn::h_centroids2);
        sendCentroToGpu(); //send newly computed centro position in phase2 to GPU

    }

    void imp_KmeansV2(){
        //initialisation
        int N = kmn::NBPOINTS;
		int M = 256;

        // phase1 assignment
        kernelAssign << <(N + M - 1) / M, M >> > (kmn::d_points, kmn::d_centroids1, kmn::NBPOINTS, kmn::NBCENTROIDS);
        checkKernelErrors();
        hipDeviceSynchronize();

        //phase 2 reduction
        N = kmn::NBCENTROIDS;
        kernelReduce << <(N + M - 1) / M, M >> >(kmn::d_points, kmn::d_centroids2, kmn::NBPOINTS, kmn::NBCENTROIDS);
        checkKernelErrors();
        hipDeviceSynchronize();
        
        getPointsFromGPU(); //fetch label
        getCentroFromGPU(); //fetch centro
        

        std::swap(kmn::d_centroids1, kmn::d_centroids2);

    }


}//end namespace gpu




namespace wdw{

    //warning, must be coherent with km::NBPOINTS and kmn::NBCENTROIDS
    static int bufferPointCount = 16*1024;
    static int bufferCentroCount = 128;

    void applyParam(){
        gbl::paused = true;
        kmn::NBPOINTS = bufferPointCount;
        kmn::NBCENTROIDS = bufferCentroCount;
        reinit();
        if(gbl::mode == GPU_MODE){
            sendPointsToGPU();
            sendCentroToGpu();
        }
        gbl::paused = false;
    }

    static void HelpMarker(const char* desc){
        ImGui::TextDisabled("(?)");
        if (ImGui::BeginItemTooltip())
        {
            ImGui::PushTextWrapPos(ImGui::GetFontSize() * 35.0f);
            ImGui::TextUnformatted(desc);
            ImGui::PopTextWrapPos();
            ImGui::EndTooltip();
        }
    }

    void kmeansParam(){
        ImGui::Begin("Kmeans");


        ImGui::SeparatorText("Advanced parameters");
        ImGui::InputInt("max iter/frame", &gbl::max_fps);

        ImGui::InputInt("points", &bufferPointCount);
        ImGui::SameLine(); HelpMarker("Number of points");

        ImGui::InputInt("clusters", &bufferCentroCount);
        ImGui::SameLine(); HelpMarker("Number of centroids (cluster)");
        if(ImGui::Button("apply")) applyParam();


        ImGui::SeparatorText("Options");
        if(ImGui::Button("randomize points")){
            gbl::paused = true;
            kmn::randomPoints();
            kmn::randomClusters();
            if(gbl::mode == GPU_MODE){
                sendPointsToGPU();
                sendCentroToGpu();
            }
            gbl::paused = false;
        }

        ImGui::DragFloat("dpos", &kmn::dpoints, 0.01f, 0.0f,3.0f, "%.3f");
        ImGui::SameLine(); HelpMarker("dispersion factor");




        ImGui::End();
    }

    void wdw_additional(){
        ImGui::SeparatorText("GPU mode");
        static int current_gpu_mode = 0;
        const char* items[] = { "version 1", "version 2"};

        if (ImGui::Combo("GPU", &current_gpu_mode, items, IM_ARRAYSIZE(items))) {
            switch (current_gpu_mode)
            {
            //save cbk for switching between modes
            case 0: 
                gpu::gpu_cbk = gpu::imp_KmeansV1; 
                getCentroFromGPU();
                break;
            case 1:
                gpu::gpu_cbk = gpu::imp_KmeansV2;
                sendCentroToGpu();
                break;
            default: break;
            }
            gbl::display = gpu::gpu_cbk;
        }
        if(current_gpu_mode == 1){
            ImGui::SameLine(); HelpMarker(
                    "Version 1 : phase 2 on CPU\n"
                    "Version 2 : phase 2 on GPU with a second kernel");
        }
    }
}//end namespace wdw




void clean(){
	switch (gbl::mode){
        case CPU_MODE: gpu::clean(); break;
        case GPU_MODE: cpu::clean(); break;
	}
}
void init(){
	switch (gbl::mode){
        case CPU_MODE: cpu::init(); break;
        case GPU_MODE: gpu::init(); break;
	}
    kmn::randomPoints();
    kmn::randomClusters();
    if(gbl::mode == GPU_MODE){
        sendPointsToGPU();
        sendCentroToGpu();
    }
}
void reinit(){
    hipDeviceSynchronize();
	switch (gbl::mode){
        case CPU_MODE: cpu::reinit(); break;
        case GPU_MODE: gpu::reinit(); break;
	}
    kmn::randomPoints();
    kmn::randomClusters();
    hipDeviceSynchronize();
}


namespace cbk{
    /*various callback
    You must ALWAYS forward the event to ImGui before processing it (except window resizing)
    You can find relevant ImGui callback in ./imgui/imgui_impl_glfw.cpp line 536 in function ImGui_ImplGlfw_InstallCallbacks
    */


    void key(GLFWwindow* window, int key, int scancode, int action, int mods){
        // Forward the event to ImGui
        ImGui_ImplGlfw_KeyCallback(window, key, scancode, action, mods);

        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureKeyboard){
            /* uses US keyboard layout ! https://www.glfw.org/docs/latest/group__keys.html
            use charCallback if you want to avoid translation qwerty->azerty*/
            if (key == GLFW_KEY_Z && action == GLFW_PRESS){ //match W in azerty
                gbl::otherWindow = !gbl::otherWindow;
            }
        }
    }

    void updt_mpos(double xpos, double ypos){
        h_params.mx = h_params.offset.a + (float)(0.003f*(xpos - gbl::SCREEN_X / 2));
        h_params.my = h_params.offset.b - (float)(0.003f*(ypos - gbl::SCREEN_Y / 2));
    }

    void mouse_button(GLFWwindow* window, int button, int action, int mods){
        // Forward the event to ImGui
        ImGui_ImplGlfw_MouseButtonCallback(window, button, action, mods);

        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            double xpos, ypos;
            glfwGetCursorPos(window, &xpos, &ypos);
            if(button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS){
                updt_mpos(xpos, ypos);
            }
            if(button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_PRESS){
                h_params.offset.a += (float)(0.003f * (xpos - gbl::SCREEN_X / 2));
		        h_params.offset.b += -(float)(0.003f * (ypos - gbl::SCREEN_Y / 2));
            }
        }
    }

    void cursor_position(GLFWwindow* window, double xpos, double ypos){
        //forward the event to ImGui
        ImGui_ImplGlfw_CursorPosCallback(window, xpos, ypos);

        //if ImGui doesn't want the event, process i
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            int leftState = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT);
            if(leftState == GLFW_PRESS){
                updt_mpos(xpos, ypos);
            }
        }
    }

    void scroll(GLFWwindow* window, double xoffset, double yoffset){
        // Forward the event to ImGui
        ImGui_ImplGlfw_ScrollCallback(window, xoffset, yoffset);

        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            float fac = (GLFW_PRESS == glfwGetKey(window, GLFW_KEY_LEFT_CONTROL)) ?  1.16f : 1.05f;
            if (yoffset >0) h_params.scale /= fac;
	        else h_params.scale *= fac;
        }
    }

    void window_size(GLFWwindow* window, int width, int height){
        //reszing logic handled in gbl::resizePixelsBuffer() called from gbl::calculate
        gbl::paused = true;
        gbl::needResize = true;
    }

}//end namespace cbk



int main(void){
    GLFWwindow* window;


    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(gbl::SCREEN_X, gbl::SCREEN_Y, TITLE, NULL, NULL);
    if (!window){
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    utl::initImGui(window);

    /* malloc values ...*/
    init();
    {/* set up generic parameters*/
        h_params.scale = 10.0f;
        h_params.mx = 0.0f;
        h_params.my = 0.0f;
        h_params.offset = Complex(0.0f, 0.0f);



        //framerate
        gbl::max_fps = 5;

        //gpu modes
        gpu::gpu_cbk = gpu::imp_KmeansV1;

        //kmn



        glClearColor(0.3,0.3,0.3,1.0);
        glColor4f(1.0,1.0,1.0,1.0);
        glDisable(GL_DEPTH_TEST);
        glPointSize(2.0f);

    }

    /* Initialize callback*/
    glfwSetKeyCallback(window, cbk::key);
    glfwSetMouseButtonCallback(window, cbk::mouse_button);
    glfwSetCursorPosCallback(window, cbk::cursor_position);
    glfwSetScrollCallback(window, cbk::scroll);
    glfwSetWindowSizeCallback(window, cbk::window_size);


    /*start timer*/
    double last_frame_time = glfwGetTime();

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Poll for and process events */
        glfwPollEvents();

        /* Interface*/
        utl::newframeImGui();
        if(gbl::otherWindow) {
            utl::wdw_info(gbl::mode, gbl::SCREEN_X,gbl::SCREEN_Y,gbl::currentFPS);
            wdw::kmeansParam();
        }

        //timer management
        double curr_time = glfwGetTime();
        double framerate = (double)1 / (double)gbl::max_fps;


        /* Render */
        gbl::calculate(window);
        gpu::setDeviceParameters(h_params);
        if(curr_time -  last_frame_time > framerate){
            gbl::display();
            last_frame_time = curr_time;
        }

        if(!gbl::paused){
            cameraApply(-h_params.mx,h_params.my,h_params.scale);
            glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

            //points
            glPointSize(1.0f);
            glEnableClientState(GL_VERTEX_ARRAY);
            glEnableClientState(GL_COLOR_ARRAY);
            glVertexPointer(3, GL_FLOAT, sizeof(Point), &(kmn::h_points->pos));
            glColorPointer(3, GL_FLOAT, sizeof(Point), &(kmn::h_points->col));
            glDrawArrays(GL_POINTS, 0, kmn::NBPOINTS);
            glDisableClientState(GL_COLOR_ARRAY);
            glDisableClientState(GL_VERTEX_ARRAY);

            //centroids
            glPointSize(3.0f);
            glColor4f(1.0f, 1.0f, 1.0f, 1.0f);
            glEnableClientState(GL_VERTEX_ARRAY);
            glVertexPointer(3, GL_FLOAT, sizeof(Point), &(kmn::h_centroids1->pos));
            glDrawArrays(GL_POINTS, 0, kmn::NBCENTROIDS);
            glDisableClientState(GL_VERTEX_ARRAY);
        }


        /* end frame for imgui*/
        utl::endframeImGui();
        utl::multiViewportImGui(window);


        /* Swap front and back buffers */
        glfwSwapBuffers(window);
    }

    utl::shutdownImGui();
    glfwTerminate();
    return 0;
}