#include "hip/hip_runtime.h"
#include "util.hpp"
#define TITLE "template"

/* FIXME add readme here*/

struct Param {
    float scale;
    float mx, my;
};
Param h_params;
/*
 You can send params to GPU with :
 __constant__ Param d_params;
 memory if you need to have per-frame constant parameter common to every pixel instead of passing them as function argument. 
 You'll need to copy h_params to d_params each frame
 Be mindful whith memory access to d_params in device functions !
 */


namespace cpu{
    //forward declaration ...
    void example();

    void init(){
        gbl::pixels = (float4*)malloc(gbl::SCREEN_X*gbl::SCREEN_Y*sizeof(float4));
        gbl::display = example;
    }
    void clean(){
        free(gbl::pixels);
    }
    void reinit(){
        gbl::pixels = (float4*)realloc(gbl::pixels, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4));
    }

    void example(){
        int i, j;
        for (i = 0; i<gbl::SCREEN_Y; i++){
            for (j = 0; j<gbl::SCREEN_X; j++){
                float x = (float)(h_params.scale*(j - gbl::SCREEN_X / 2));
                float y = (float)(h_params.scale*(i - gbl::SCREEN_Y / 2));
                float4* p = gbl::pixels + (i*gbl::SCREEN_X + j);
                // default: black
                p->x = 0.0f;
                p->y = 0.0f;
                p->z = 0.0f;
                p->w = 1.0f;
                if (sqrt((x - h_params.mx)*(x - h_params.mx) + (y - h_params.my)*(y - h_params.my))<0.01)
                    p->x = 1.0f;
                else if ((i == gbl::SCREEN_Y / 2) || (j == gbl::SCREEN_X / 2))
                {
                    p->x = 1.0f;
                    p->y = 1.0f;
                    p->z = 1.0f;
                }
            }
        }
    }

}//end namespace cpu

namespace gpu{ //replace with your own code
    //forward declaration ...
    void imp_Julia();

    void init(){
        gbl::pixels = (float4*)malloc(gbl::SCREEN_X*gbl::SCREEN_Y*sizeof(float4));
        gbl::display = cpu::example;
    }
    void reinit(){
        gbl::pixels = (float4*)realloc(gbl::pixels, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4));
    }
    void clean(){
        free(gbl::pixels);
    }
    
}//end namespace gpu

void clean(){
	switch (gbl::mode){
        case CPU_MODE: gpu::clean(); break;
        case GPU_MODE: cpu::clean(); break;
	}
}
void init(){
	switch (gbl::mode){
        case CPU_MODE: cpu::init(); break;
        case GPU_MODE: gpu::init(); break;
	}
}
void reinit(){
	switch (gbl::mode){
        case CPU_MODE: cpu::reinit(); break;
        case GPU_MODE: gpu::reinit(); break;
	}
}



namespace cbk{ 
    /*various callback
    You must ALWAYS forward the event to ImGui before processing it (except window resizing)
    You can find relevant ImGui callback in ./imgui/imgui_impl_glfw.cpp line 536 in function ImGui_ImplGlfw_InstallCallbacks
    */

   void key(GLFWwindow* window, int key, int scancode, int action, int mods){
        // Forward the event to ImGui
        ImGui_ImplGlfw_KeyCallback(window, key, scancode, action, mods);


        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureKeyboard){
            /* uses US keyboard layout ! https://www.glfw.org/docs/latest/group__keys.html
            use charCallback if you want to avoid translation qwerty->azerty*/
            if (key == GLFW_KEY_Z && action == GLFW_PRESS){ //match W in azerty
                gbl::otherWindow = !gbl::otherWindow;
            }
        }
    }

    inline void updt_mpos(double xpos, double ypos){
        h_params.mx = (float)(h_params.scale*(xpos - gbl::SCREEN_X / 2));
        h_params.my = - (float)(h_params.scale*(ypos - gbl::SCREEN_Y / 2));
    }

    void mouse_button(GLFWwindow* window, int button, int action, int mods){
        // Forward the event to ImGui
        ImGui_ImplGlfw_MouseButtonCallback(window, button, action, mods);
        
        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            double xpos, ypos;
            glfwGetCursorPos(window, &xpos, &ypos);
            if(button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS){
                updt_mpos(xpos, ypos);
            }
            if(button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_PRESS){
                //...
            }
        }
    }

    void cursor_position(GLFWwindow* window, double xpos, double ypos){
        //forward the event to ImGui
        ImGui_ImplGlfw_CursorPosCallback(window, xpos, ypos);

        //if ImGui doesn't want the event, process i
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            int leftState = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT);
            if(leftState == GLFW_PRESS){
                h_params.mx = (float)(h_params.scale*(xpos - gbl::SCREEN_X / 2));
                h_params.my = -(float)(h_params.scale*(ypos - gbl::SCREEN_Y / 2));
            }
        }
    }

    void scroll(GLFWwindow* window, double xoffset, double yoffset){
        // Forward the event to ImGui
        ImGui_ImplGlfw_ScrollCallback(window, xoffset, yoffset);
        
        //if ImGui doesn't want the event, process it
        ImGuiIO& io = ImGui::GetIO();
        if(!io.WantCaptureMouse){
            if (yoffset >0) h_params.scale /= 1.05f;
	        else h_params.scale *= 1.05f;
        }
    }

    void window_size(GLFWwindow* window, int width, int height){
        //reszing logic handled in gbl::resizePixelsBuffer() called from gbl::calculate
        gbl::needResize = true;
        gbl::paused = true;
    }

}//end namespace cbk

int main(void){
    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(gbl::SCREEN_X, gbl::SCREEN_Y, TITLE, NULL, NULL);
    if (!window){
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    utl::initImGui(window);

    /* malloc render specific arrays... */
    init();
    {/* set up parameter*/
        h_params.scale = 0.003f;
        h_params.mx = 0.0f;
        h_params.my = 0.0f;

        //add your own parameters ...
    }

    /* Initialize callback*/
    glfwSetKeyCallback(window, cbk::key);
    glfwSetMouseButtonCallback(window, cbk::mouse_button);
    glfwSetCursorPosCallback(window, cbk::cursor_position);
    glfwSetScrollCallback(window, cbk::scroll);
    glfwSetWindowSizeCallback(window, cbk::window_size);


    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Poll for and process events */
        glfwPollEvents();

        /* Interface*/
        utl::newframeImGui();
        if(gbl::otherWindow) utl::wdw_info(gbl::mode, gbl::SCREEN_X,gbl::SCREEN_Y,gbl::currentFPS);
        
        /* Render here */
        gbl::calculate(window);
        gbl::display();
        if(!gbl::paused) glDrawPixels(gbl::SCREEN_X, gbl::SCREEN_Y, GL_RGBA, GL_FLOAT, gbl::pixels);
        
        /* Interface */
        utl::endframeImGui();
        utl::multiViewportImGui(window);
        
        /* Swap front and back buffers */
        glfwSwapBuffers(window);
    }

    utl::shutdownImGui();
    glfwTerminate();
    return 0;
}