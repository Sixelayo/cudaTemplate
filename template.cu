#include "hip/hip_runtime.h"
#include "util.hpp"
#define TITLE "template"

/* use task to compile or run command TODO*/



namespace prm{

    float scale = 0.003f;
    //mouse coordinate
    float mx, my;

} //end namespace prm


namespace cpu{
    //forward declaration ...
    void example(); void imp_Julia();

    void init(){
        gbl::pixels = (float4*)malloc(gbl::SCREEN_X*gbl::SCREEN_Y*sizeof(float4));
        gbl::display = example;
    }
    void reinit(){
        gbl::pixels = (float4*)realloc(gbl::pixels, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4));
    }
    void clean(){
        free(gbl::pixels);
    }

    void example(){
        int i, j;
        for (i = 0; i<gbl::SCREEN_Y; i++){
            for (j = 0; j<gbl::SCREEN_X; j++){
                float x = (float)(prm::scale*(j - gbl::SCREEN_X / 2));
                float y = (float)(prm::scale*(i - gbl::SCREEN_Y / 2));
                float4* p = gbl::pixels + (i*gbl::SCREEN_X + j);
                // default: black
                p->x = 0.0f;
                p->y = 0.0f;
                p->z = 0.0f;
                p->w = 1.0f;
                if (sqrt((x - prm::mx)*(x - prm::mx) + (y - prm::my)*(y - prm::my))<0.01)
                    p->x = 1.0f;
                else if ((i == gbl::SCREEN_Y / 2) || (j == gbl::SCREEN_X / 2))
                {
                    p->x = 1.0f;
                    p->y = 1.0f;
                    p->z = 1.0f;
                }
            }
        }
    }

}//end namespace cpu

namespace gpu{
    //forward declaration ...
    void imp_Julia();

    void init(){
        gbl::pixels = (float4*)malloc(gbl::SCREEN_X*gbl::SCREEN_Y*sizeof(float4));
        gbl::display = cpu::example;
    }
    void reinit(){
        gbl::pixels = (float4*)realloc(gbl::pixels, gbl::SCREEN_X * gbl::SCREEN_Y * sizeof(float4));
    }
    void clean(){
        free(gbl::pixels);
    }
    
}//end namespace gpu

void clean(){
	switch (gbl::mode){
        case CPU_MODE: gpu::clean(); break;
        case GPU_MODE: cpu::clean(); break;
	}
}
void init(){
	switch (gbl::mode){
        case CPU_MODE: cpu::init(); break;
        case GPU_MODE: gpu::init(); break;
	}
}
void reinit(){
	switch (gbl::mode){
        case CPU_MODE: cpu::reinit(); break;
        case GPU_MODE: gpu::reinit(); break;
	}
}


namespace gbl{
    void resizePixelsBuffer(){
        reinit();
    }

    //handle fps computation, and reallocating buffer if needed(to avoid too many call to mallloc/free)
    void calculate(GLFWwindow* window){
        frameAcc++;
        double timeCurr  = glfwGetTime();
        float elapsedTime = timeCurr-prevUpdt;
        if(elapsedTime>FPS_UPDATE_DELAY){
            currentFPS = frameAcc / elapsedTime ;
            frameAcc = 0;
            prevUpdt = timeCurr;
            if(needResize){
                glfwGetWindowSize(window, &SCREEN_X, &SCREEN_Y);
                resizePixelsBuffer();
                paused = false;
                needResize = false;
            }
        }

    }
}


namespace cbk{ 
    /*various callback
    You must ALWAYS forward the event to ImGui before processing it (except window resizing)
    You can find relevant ImGui callback in ./imgui/imgui_impl_glfw.cpp line 536 in function ImGui_ImplGlfw_InstallCallbacks
    */

    // void mouse_button(GLFWwindow* window, int button, int action, int mods){
    //     // Forward the event to ImGui
    //     ImGuiIO& io = ImGui::GetIO();
    //     ImGui_ImplGlfw_MouseButtonCallback(window, button, action, mods);
        
    //     //if ImGui doesn't want the event, process it
    //     if(!io.WantCaptureMouse){
            
    //     }
    // }

    static void cursor_position(GLFWwindow* window, double xpos, double ypos){
        //forward the event to ImGui
        ImGuiIO& io = ImGui::GetIO();
        ImGui_ImplGlfw_CursorPosCallback(window, xpos, ypos);

        //if ImGui doesn't want the event, process i
        if(!io.WantCaptureMouse){
            int leftState = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT);
            if(leftState == GLFW_PRESS){
                prm::mx = (float)(prm::scale*(xpos - gbl::SCREEN_X / 2));
                prm::my = -(float)(prm::scale*(ypos - gbl::SCREEN_Y / 2));
            }
        }
    }

    void scroll(GLFWwindow* window, double xoffset, double yoffset){
        // Forward the event to ImGui
        ImGuiIO& io = ImGui::GetIO();
        ImGui_ImplGlfw_ScrollCallback(window, xoffset, yoffset);
        
        //if ImGui doesn't want the event, process it
        if(!io.WantCaptureMouse){
            if (yoffset >0) prm::scale /= 1.05f;
	        else prm::scale *= 1.05f;
        }
    }

    void window_size(GLFWwindow* window, int width, int height){
        //reszing logic handled in gbl::resizePixelsBuffer() called from gbl::calculate
        gbl::needResize = true;
        gbl::paused = true;
    }

}//end namespace cbk

int main(void){
    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(gbl::SCREEN_X, gbl::SCREEN_Y, TITLE, NULL, NULL);
    if (!window){
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    utl::initImGui(window);

    /* init render specific values*/
    init();

    /* Initialize callback*/
    //glfwSetMouseButtonCallback(window, cbk::mouse_button);
    glfwSetCursorPosCallback(window, cbk::cursor_position);
    glfwSetScrollCallback(window, cbk::scroll);
    glfwSetWindowSizeCallback(window, cbk::window_size);


    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Poll for and process events */
        glfwPollEvents();

        /* Interface*/
        utl::newframeImGui();
        if(gbl::otherWindow) utl::wdw_info(gbl::mode, gbl::SCREEN_X,gbl::SCREEN_Y,gbl::currentFPS);
        
        /* Render here */
        gbl::calculate(window);
        gbl::display();
        if(!gbl::paused) glDrawPixels(gbl::SCREEN_X, gbl::SCREEN_Y, GL_RGBA, GL_FLOAT, gbl::pixels);
        
        /* end frame for imgui*/
        utl::endframeImGui();
        utl::multiViewportImGui(window);
        

        /* Swap front and back buffers */
        glfwSwapBuffers(window);
    }

    utl::shutdownImGui();
    glfwTerminate();
    return 0;
}